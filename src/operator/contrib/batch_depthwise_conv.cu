#include "hip/hip_runtime.h"
#include "./batch_depthwise-inl.h"
#include "./batch_depthwise.cuh"
#include <vector>


namespace mxnet {
namespace op {


template<typename DType>
void BatchDW2dForwardGpu(mshadow::Stream<gpu> *stream,
                         const DepthwiseArgs& args,
                         const std::vector<TBlob> &in_data,
                         const std::vector<TBlob> &out_data) {
  using namespace mshadow;
  using namespace mshadow::expr;
  using namespace tf::depthwise_conv;
  using namespace tf::depthwise_conv::cuda;
  Tensor<gpu, 4, DType> data = in_data[bdw::kData].get<gpu, 4, DType>(stream);
  Tensor<gpu, 4, DType> weight = in_data[bdw::kWeight].get<gpu, 4, DType>(stream);
  Tensor<gpu, 4, DType> out = out_data[bdw::kOut].get<gpu, 4, DType>(stream);
  int num_output = out_data[bdw::kOut].shape_.Size();
  int block_num = std::min(num_output/mshadow::cuda::kBaseThreadNum + 1,
    mshadow::cuda::kMaxGridNum);
  auto s = mshadow::Stream<gpu>::GetStream(stream);
  DepthwiseConv2dForwardKernel<DType, -1, -1>
          <<<block_num, mshadow::cuda::kBaseThreadNum, 0, s>>>(data.dptr_,
                                                               weight.dptr_,
                                                               args,
                                                               num_output,
                                                               out.dptr_);
  MSHADOW_CUDA_POST_KERNEL_CHECK(DepthwiseConv2dForwardKernel);
}


template<typename DType>
void DepthwiseConv2dBackwardDataGpu(mshadow::Stream<gpu> *stream,
                                    const DepthwiseArgs& args,
                                    const std::vector<TBlob> &out_grad,
                                    const std::vector<TBlob> &in_data,
                                    const std::vector<TBlob> &in_grad) {
  using namespace mshadow;
  using namespace mshadow::expr;
  using namespace tf::depthwise_conv;
  using namespace tf::depthwise_conv::cuda;
  Tensor<gpu, 4, DType> out_g = out_grad[bdw::kOut].get<gpu, 4, DType>(stream);
  Tensor<gpu, 4, DType> weight = in_data[bdw::kWeight].get<gpu, 4, DType>(stream);
  Tensor<gpu, 4, DType> in_data_g = in_grad[bdw::kData].get<gpu, 4, DType>(stream);
  // select kernel

  int num_in_grad = in_grad[bdw::kData].shape_.Size();
  auto s = mshadow::Stream<gpu>::GetStream(stream);
  int block_num = std::min(num_in_grad/mshadow::cuda::kBaseThreadNum + 1,
                             mshadow::cuda::kMaxGridNum);
  DepthwiseConv2dBackwardDataKernel<DType>
        <<<block_num, mshadow::cuda::kBaseThreadNum, 0, s>>>(args,
                                                             out_g.dptr_,
                                                             weight.dptr_,
                                                             in_data_g.dptr_,
                                                             num_in_grad);
  MSHADOW_CUDA_POST_KERNEL_CHECK(DepthwiseConv2dBackwardDataKernel);
}

template<typename DType>
void DepthwiseConv2dBackwardFilterGpu(mshadow::Stream<gpu> *stream,
                                      const DepthwiseArgs& args,
                                      const std::vector<TBlob> &out_grad,
                                      const std::vector<TBlob> &in_data,
                                      const std::vector<TBlob> &in_grad) {
  using namespace mshadow;
  using namespace mshadow::expr;
  using namespace tf::depthwise_conv;
  using namespace tf::depthwise_conv::cuda;
  Tensor<gpu, 4, DType> out_g = out_grad[bdw::kOut].get<gpu, 4, DType>(stream);
  Tensor<gpu, 4, DType> in_d = in_data[bdw::kData].get<gpu, 4, DType>(stream);
  Tensor<gpu, 4, DType> weight_grad = in_grad[bdw::kWeight].get<gpu, 4, DType>(stream);
  // select kernel
    int num_out_grad = out_grad[conv::kOut].shape_.Size();
    auto s = mshadow::Stream<gpu>::GetStream(stream);
    int block_num = std::min(args.out_channel * args.batch, mshadow::cuda::kMaxGridNum);

    DepthwiseConv2dBackwardFilterKernel<DType, -1, -1>
          <<<block_num, mshadow::cuda::kBaseThreadNum, 0, s>>>(args,
                                                               out_g.dptr_,
                                                               in_d.dptr_,
                                                               weight_grad.dptr_,
                                                               num_out_grad);
    MSHADOW_CUDA_POST_KERNEL_CHECK(DepthwiseConv2dBackwardFilterKernel);
  }
}



template<>
void BatchDWCompute<gpu>(const nnvm::NodeAttrs& attrs,
                               const OpContext& ctx,
                               const std::vector<TBlob>& inputs,
                               const std::vector<OpReqType>& req,
                               const std::vector<TBlob>& outputs) {
  const BatchDWParam& param = nnvm::get<BatchDWParam>(attrs.parsed);
  mxnet::ShapeVector in_shape(inputs.size());
  mxnet::ShapeVector out_shape(1, outputs[0].shape_);
  for (size_t i = 0; i < in_shape.size(); i++)
    in_shape[i] = inputs[i].shape_;
  int dtype = inputs[bdw::kData].type_flag_;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    BatchDWOp<gpu, DType> op;
    op.Init(param, in_shape, out_shape);
    // op.Forward(ctx, inputs, req, outputs);
    auto stream = ctx.get_stream<gpu>();
    CHECK_EQ(req[bdw::kOut], kWriteTo);
    BatchDW2dForwardGpu<float>(stream, op.args_, inputs, outputs);
  })
}

template<>
void BatchDWGradCompute<gpu>(const nnvm::NodeAttrs& attrs,
                                 const OpContext& ctx,
                                 const std::vector<TBlob>& inputs,
                                 const std::vector<OpReqType>& req,
                                 const std::vector<TBlob>& outputs) {
  const BatchDWParam& param = nnvm::get<BatchDWParam>(attrs.parsed);
  std::vector<TBlob> in_data(inputs.begin() + 1, inputs.end());
  const TBlob &out_grad = inputs[0];
  const std::vector<TBlob> &in_grad = outputs;
  int dtype = out_grad.type_flag_;
  mxnet::ShapeVector in_shape(in_data.size());
  mxnet::ShapeVector out_shape(1, out_grad.shape_);
  for (size_t i = 0; i < in_shape.size(); i++)
    in_shape[i] = in_data[i].shape_;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    BatchDWOp<gpu, DType> op;
    op.Init(param, in_shape, out_shape);
    // op.Backward(ctx, std::vector<TBlob>{out_grad}, in_data, req, in_grad);
    auto stream = ctx.get_stream<gpu>();
    // data
    if (req[bdw::kData] != kNullOp) {
      if (req[bdw::kData] != kAddTo) {
        mshadow::Tensor<gpu, 4, DType> igrad = in_grad[bdw::kData].get<gpu, 4, DType>(stream);
        igrad = 0.0f;
      }
      DepthwiseConv2dBackwardDataGpu<float>(stream,
                                     op.args_,
                                     std::vector<TBlob>{out_grad},
                                     in_data, in_grad);

    }
    if (req[bdw::kWeight] != kNullOp) {
      if (req[bdw::kWeight] != kAddTo) {
        mshadow::Tensor<gpu, 4, DType> wgrad = in_grad[bdw::kWeight].get<gpu, 4, DType>(stream);
        wgrad = 0.0f;
      }
      DepthwiseConv2dBackwardFilterGpu<float>(stream,
        args_,
        out_grad,
        in_data,
        in_grad);
    }
  })
}


NNVM_REGISTER_OP(BatchDW)
.set_attr<FCompute>("FCompute<gpu>", BatchDWCompute<gpu>);
  
NNVM_REGISTER_OP(_backward_BatchDW)
.set_attr<FCompute>("FCompute<gpu>", BatchDWGradCompute<gpu>);

}  // namespace op
}  // namespace mxnet